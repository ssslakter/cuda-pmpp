#include "hip/hip_runtime.h"
#include <utils.cuh>

#define BLOCK_DIM 1024

__global__ void SharedMemoryReduction(float* input, float* output, int n) {
    __shared__ float input_s[BLOCK_DIM]; 
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x; // index within a block
    unsigned int t = threadIdx.x; // global index

    // Load elements into shared memory
    if (idx < n) {
        input_s[t] = input[idx];
    } else {
        input_s[t] = 0.0f;
    }
    __syncthreads();

    // Reduction in shared memory
    for (unsigned int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (t < stride && idx + stride < n) {
            input_s[t] += input_s[t + stride];
        }
        __syncthreads();
    }

    // Reduction across blocks in global memory
    // needs to be atomic to avoid contention
    if (t == 0) {
        atomicAdd(output, input_s[0]);
    }
}

torch::Tensor sum(torch::Tensor m)
{
    CHECK_INPUT(m);
    auto m = m.flatten();
    int n = m.size(0);
    auto output = torch::zeros(n, m.options());

    SharedMemoryReduction<<<1, n/2>>>(m.data_ptr<float>(), output.data_ptr<float>(), n);
    C10_CUDA_KERNEL_LAUNCH_CHECK();
    return output;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m)
{
    m.def("sum", &sum, "sum of elements in a tensor");
}
